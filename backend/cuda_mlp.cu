#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <cuda_mlp.h>

using namespace std;

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__device__ float sigmoid(float x) {
	return 1.0 / (1 + exp(0 - x));
}

__global__ void MatMulKernel(Matrix weight, float *input) {
	int input_size = weight.height;
	int output_size = weight.width;

	int tid = threadIdx.x;
	// load input into shared memory.
	//__shared__ float share_input[1024];
	extern __shared__ float share_input[];
	for (int i = tid; i < input_size - 1; i += blockDim.x) {
		share_input[i] = input[i];
	}
	__syncthreads();
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < output_size) {
		float sum = 0;
		for (int i = 0; i < input_size; i++) {
			float* row = weight.elements + i * output_size;
			if (i != input_size - 1) {
				sum += share_input[i] * row[idx];
			} else {
				sum += row[idx];
			}
		}
		input[idx] = sigmoid(sum);
	}
}


cuda_mlp::cuda_mlp() {
	weight_list = NULL;
	dev_weight = NULL;
	dev_input = NULL;
	layers_num = 0;
	max_nodes_num = 0;
	kernel_time = 0;
	load_time = 0;
	//cout << "class cuda_mlp is created" << endl;
}

int cuda_mlp::load(const char* file_path){
	//read files;
	ifstream fin(file_path);
	if(fin){
		fin>>layers_num;

		int *nodes_num = new int[layers_num];

		for (int i = 0; i < layers_num + 1; i++) {
			fin >> nodes_num[i];
			if (nodes_num[i] > max_nodes_num)
				max_nodes_num = nodes_num[i];
		}

		weight_list = new Matrix[layers_num];

		for (int i = 0; i < layers_num; i++) {

			weight_list[i].width = nodes_num[i + 1];
			weight_list[i].height = nodes_num[i] + 1;

			int num = weight_list[i].width * weight_list[i].height;
			weight_list[i].elements = new float[num];

			for (int j = 0; j < num; j++) {
				fin >> weight_list[i].elements[j];
			}
		}
	//  cout << "layers_num : " << layers_num << endl;
		delete[] nodes_num;
	}
	else{
		cout<<"can't open mod file"<<endl;
	}
	//malloc for gpu;
	 dev_weight = new Matrix[layers_num];
	for(int i=0;i<layers_num;i++){

		dev_weight[i].width = weight_list[i].width;

		dev_weight[i].height = weight_list[i].height;

		CUDA_CHECK_RETURN(
			hipMalloc((void ** )&dev_weight[i].elements,
				sizeof(float) * weight_list[i].width*weight_list[i].height));
		CUDA_CHECK_RETURN(
			hipMemcpy(dev_weight[i].elements, weight_list[i].elements, sizeof(float) * weight_list[i].width * weight_list[i].height,
				hipMemcpyHostToDevice));

	}
	CUDA_CHECK_RETURN(
		hipMalloc((void ** )&dev_input,
			sizeof(float) * max_nodes_num));
	return 0;
}

int cuda_mlp::run(const float* input, float* output){
	int output_length = dev_weight[layers_num - 1].width;
	int input_length = dev_weight[0].height - 1;
	CUDA_CHECK_RETURN(
		hipMemcpy(dev_input, input, sizeof(float) * input_length,
			hipMemcpyHostToDevice));
	for (int i = 0; i < layers_num ; i++) {
		int blockCount = (dev_weight[i].width + BLOCK_SIZE - 1)
					/ BLOCK_SIZE;
		MatMulKernel<<<blockCount, BLOCK_SIZE,1024>>>(dev_weight[i], dev_input);
	//used for debug;
	}
	CUDA_CHECK_RETURN(
		hipMemcpy(output, dev_input, sizeof(float) * output_length,
			hipMemcpyDeviceToHost));
	return 0;
	
}

void cuda_mlp::kernel_free(){
	CUDA_CHECK_RETURN(hipFree(dev_input));
	for (int i = 0; i < layers_num; i++) {
		delete[] weight_list[i].elements;
		CUDA_CHECK_RETURN(hipFree(dev_weight[i].elements));
	}
	delete[] weight_list;
	delete[] dev_weight;
}

cuda_mlp::~cuda_mlp(){
	//cout << "class cuda_mlp is deleted" << endl;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
